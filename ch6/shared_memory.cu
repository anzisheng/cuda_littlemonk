#include "hip/hip_runtime.h"
#include <cstdio>
#include <stdio.h>
#include <math.h>
//                     
//                   b00 b01 b02 b03
//                   b10 b11 b12 b13
//                   b20 b21 b22 b23
//                   b30 b32 b32 b33
//
//a00 a01 a02 a03    c00 c01 c02 c03
//a10 a11 a12 a13    c10 c12 c13 c14
//a20 a21 a22 a23    c20 c21 c22 c23
//a30 a31 a32 a33    c30 c31 c32 c33
//
//c21 = a20 *b01 + a21*b11 + c21 = c20 *b01 + c21*b11 + aa

__golbal__ void gpu_matrix(int* a, int* b, int* c, int M, int N, int K);
 void cpu_matrix(int* a, int *b, int * c, int M, int N, int K);
#define M 1000
#define N 500
#define K 1000

#define BLOCK_SIZE 16

__managed__ int a[M*N];
__managed__ int b[N*K];
__managed__ int c_gpu[M*K];
__managed__ int c_cpu[M*K];
__global__ void gpu_matrix(int* a, int* b, int* c_gpu, int m, int n, int k)
{
    __shared__ int sub_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int sub_b[BLOCK_SIZE][BLOCK_SIZE];
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int tmp = 0;
    int idx = 0;
    for(int step = 0; step < N/BLOCK_SIZE; step++)
    {
        int step_x =  step * BLOCK_SIZE + threadIdx.x;
        int step_y = y;
        idx = step_y * N + step_x;
        if(step_x > N || step_y > M)
        {
            sub_a[threadIdx.y][threadIdx.x] = 0;
        }
        else {
            sub_a[threadIdx.y][threadIdx.x] = a.[idx];

        }
        step_x = x;
        step_y = step * BLOCK_SIZE+ threadIdx.y;
        idx = step*k+step_x;
        
    }

}
void cpu_matrix(int* a, int*b, int* c_cpu, int m, int n, int k )
{
    for(int y = 0; y < m; y++)
    {
        for(int x = 0; x < k; x++)
        {
            int  temp = 0;
            for(int step = 0; step < n; step++)
            {
                temp+= a[y*n + step] * b[step*k+ x];

            } 
            c_cpu[y*k + x] = temp;
        }
    }


}


 int main()
 {
    for(int y=0; y < M ; y++)
    {
        for(int x= 0; x <N; x++)
        {
            a[y*N+x] = rand()%1024;
        }
    }

    for(int y=0; y < N; y++)
    {
        for(int x = 0; x < K; x++)
        b[y*K+x] = rand()%1024;
    }

// define thread dimensions
    unsigned int grid_x = (K + BLOCK_SIZE-1)/BLOCK_SIZE;
    unsigned int grid_y = (M + BLOCK_SIZE-1)/BLOCK_SIZE;

    dim3 dimGrid(grid_x, grid_y);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    gpu_matrix(a, b, c_gpu, M, N, K);
    cpu_matrix(a, b, c_cpu, M,N,K);
    bool errors = false;
    for(int y = 0; y < M; y++)
    {
        for(int x = 0; x < K; x++)
        {
            if(fabs(c_cpu[y*K +x] - c_gpu[y*K+x]) > 1.0e-10)
            {
                errors = true;

            }
        }
    }
    printf("result: %s \n", errors?"errors":"passed");

    return 0; 
    
    
 }